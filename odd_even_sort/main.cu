#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>
#include <string>
#define THREAD_PER_BLOCK 8

  struct mv {

    int vertex_id;
    int source_partition;
    int des_partition;
    int gain;

    mv() = default;

    __host__ __device__
    mv(int vertex_id,
              int source_partition,
              int des_partition,
              int gain) :
              vertex_id(vertex_id),
              source_partition(source_partition),
              des_partition(source_partition),
              gain(gain) {};
  };

__device__
  void swap(mv* d_data, int idx_1, int idx_2) {
    mv tmp = d_data[idx_1];
    d_data[idx_1] = d_data[idx_2];
    d_data[idx_2] = tmp;
  }

//__global__
  //void even_sort(int* d_data, int N) {
    //int gid = blockIdx.x * blockDim.x + threadIdx.x;
    //int idx = gid * 2;
    //if(idx <= (N - 2)) {
      //swap(d_data, idx, idx+1);
    //}
  //}

//__global__
  //void odd_sort(int* d_data, int N) {
    //int gid = blockIdx.x * blockDim.x + threadIdx.x;
    //int idx = gid * 2 + 1;
    //if(idx <= (N - 2)) {
      //swap(d_data, idx, idx+1);
    //}
  //}

__global__ 
  void odd_even_kernel(mv* d_data, int start_idx, int N, int num_thread_need) {
    int gid = blockIdx.x * blockDim.x + threadIdx.x;

    if(gid < num_thread_need) {
      int idx = 2 * gid + start_idx;
      if(idx < (N - 1) && d_data[idx].gain > d_data[idx + 1].gain) {
        swap(d_data, idx, idx + 1);
      }
    }
  }

  void odd_even_sort(mv* h_data, int N) {
    hipError_t cudaStatus;
    mv* d_data;
    hipMalloc((void**)&d_data, sizeof(mv) * N);
    hipMemcpy(d_data, h_data, sizeof(mv) * N, hipMemcpyHostToDevice);
    //int num_thread_need = (N - 1) / 2 + (N - 1) % 2;
    int num_thread_need = N / 2;
    std::cout << "num_thread_need: " << num_thread_need << '\n';
    int num_block = (num_thread_need  + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK;
    for(int i = 0; i < N; i ++) {
      //odd_even_kernel<<< num_block, std::min(num_thread_need, THREAD_PER_BLOCK) >>> (d_data, i % 2, N, num_thread_need); 
      odd_even_kernel<<< num_block, THREAD_PER_BLOCK >>> (d_data, i % 2, N, num_thread_need); 
    }
    hipMemcpy(h_data, d_data, sizeof(mv) * N, hipMemcpyDeviceToHost);
    hipFree(d_data);
  }

  //void odd_even_sort(int* h_data, int N) {
    //cudaError_t cudaStatus;
    //int* d_data;
    //cudaMalloc((void**)&d_data, sizeof(int) * N);
    //cudaMemcpy(d_data, h_data, sizeof(int) * N, cudaMemcpyHostToDevice);
    ////int num_thread_need = (N - 1) / 2 + (N - 1) % 2;
    //int num_block = (N  + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK;
    //for(int i = 0; i <= N/2; i ++) {
      //even_sort<<< num_block, THREAD_PER_BLOCK >>> (d_data, N); 
      //odd_sort<<< num_block, THREAD_PER_BLOCK >>> (d_data, N); 
    //}
    //cudaMemcpy(h_data, d_data, sizeof(int) * N, cudaMemcpyDeviceToHost);
    //cudaFree(d_data);
  //}

  int main(int argc, char** argv) {

    //std::vector<int> h_in = {1,5,3,2,4,7,8,11,6,3,12,4,1,6,8};
    const int N = std::stoi(argv[1]);
    std::vector<mv> h_in = {};

    for(int i = 0 ; i < N; i++) {
      mv mv_request;
      mv_request.vertex_id = i;
      mv_request.source_partition = rand()%4;
      mv_request.des_partition = rand()%4;
      mv_request.gain = rand()%15;
      h_in.push_back(mv_request);
    }
 
    for(int i = 0; i < h_in.size(); i++) {
      std::cout << "before sort, i: " << i << '\n';
      std::cout << "mv.vertex_id: " << i << ", mv.source_partition: " << h_in[i].source_partition << ", des_partition: " << h_in[i].des_partition << ", mv.gain: " << h_in[i].gain << '\n';
      std::cout << "-----------------------------\n";
    }
    odd_even_sort(h_in.data(), h_in.size());
    for(int i = 0; i < h_in.size(); i++) {
      std::cout << "after sort, i: " << i << '\n';
      std::cout << "mv.vertex_id: " << i << ", mv.source_partition: " << h_in[i].source_partition << ", des_partition: " << h_in[i].des_partition << ", mv.gain: " << h_in[i].gain << '\n';
      std::cout << "-----------------------------\n";
    }
    return 0;
  }
