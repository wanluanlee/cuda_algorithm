
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

//__global__
//void scan(int* d_in, int* d_out, int N) {

  //extern __shared__ int temp[];
  //unsigned tid = threadIdx.x;
  //if(tid > N) {
    //return;
  //}
  //int pout = 0; 
  //int pin = 1;
  //temp[tid*N+tid] = (tid > 0) ? d_in[tid-1] : 0;

  //__syncthreads();
  //if(threadIdx.x == 0) {
    //for(int i = 0; i <= N; i++) {
      //printf("i:%d, temp:%d \n", i, temp[i]);
 
    //}

  //}

  //__syncthreads();
  //for(int offset = 1; offset < N; offset *= 2) {
    //pout = 1 - pout;
    //pin = 1 - pout;

    //if(tid >= offset) {
      //temp[pout*N+tid] += temp[pin*N+tid - offset];
      //printf("tid:%d, pout:%d, pin:%d, idx_left:%d, idx_right:%d, offset:%d \n", tid, pout, pin, pout*N+tid, pin*N+tid-offset, offset);
    //}
    //else {
      //temp[pout*N+tid] = temp[pin*N+tid];
      //printf(" less offset, tid:%d, pout:%d, pin:%d, idx_left:%d, idx_right:%d, offset:%d \n", tid, pout, pin, pout*N+tid, pin*N+tid-offset, offset);
    //}
    //__syncthreads();
  //}

  //d_out[tid] = temp[pout*N+tid];
//}

__global__
void scan(int* d_in, int* d_out, int N) {
  int tmp;
  for(int off = 1; off < N; off *= 2) {
    if(threadIdx.x >= off) {
      tmp = d_in[threadIdx.x - off];
      //printf("in idx:%d, d_in:%d \n", threadIdx.x - off, d_in[threadIdx.x-off]);
    }
    __syncthreads();
    if(threadIdx.x >= off) {
      d_in[threadIdx.x] += tmp;
      //printf("write to out idx:%d \n", threadIdx.x);
    }
    __syncthreads();
  }

}

int main() {

 std::vector<int> h_in = {0,2,4,6,7,12,14};
 const int N = h_in.size();
 int* d_in;
 int* d_out;
 int* h_out = (int*) malloc(sizeof(int)*N);
 hipMalloc((void**)&d_in, sizeof(int) * N);
 hipMalloc((void**)&d_out, sizeof(int) * N);
 hipMemcpy(d_in, h_in.data(), sizeof(int)*N, hipMemcpyHostToDevice);

 scan <<< 1, N>>> (d_in, d_out, N);
 //cudaDeviceSynchronize();

 hipMemcpy(h_out, d_in, sizeof(int)*N, hipMemcpyDeviceToHost);
 for(int i = 0; i < N; i++) {
   std::cout << "i:" << i << ", val: " << h_out[i] << '\n';
 }
 hipFree(d_in);
 hipFree(d_out);
 free(h_out);
  return 0;
}
