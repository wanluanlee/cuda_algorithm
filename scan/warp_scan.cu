
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <numeric>
#define FULL_MASK 0xffffffff

__global__
void scan(int* d_in, int N) {
  //unsigned mask = __ballot_sync(FULL_MASK, threadIdx.x < N);
  int tid = threadIdx.x;
  __shared__ int tmp[32];
  int tmp1, tmp2, tmp3;
  if(threadIdx.x > N) {
    return;
  }
  //if(threadIdx.x < N) {
    tmp1 = d_in[threadIdx.x];
    for(int off = 1; off < N; off *= 2) {
      tmp2 = __shfl_up_sync(FULL_MASK, tmp1, off);
      if(threadIdx.x % 32 >= off) {
        tmp1 += tmp2;
      }
    }
    if(tid % 32 == 31) {
      tmp[tid/32] = tmp1;
    }
    __syncthreads();
    if(threadIdx.x == 0) {
      for(int i = 0; i < 32; i++) {
        printf("i:%d, share val:%d \n", i, tmp[i]);
      }
    }
    __syncthreads();
    //printf("threadIdx.x:%d, tmp1:%d \n", threadIdx.x, tmp1);
  //}
  if(tid < 32) {
    tmp2 = 0.0f;
    if(tid < blockDim.x/32) {
      tmp2 = tmp[tid];
      printf("tmp2:%d, tid:%d, blockDim.x:%d \n", tmp2, tid, blockDim.x);
    }
    for(int off = 1; off < 32; off <<=1) {
      tmp3 = __shfl_up_sync(FULL_MASK, tmp2, off);
      printf("tmp3:%d, tmp2:%d, tid:%d \n", tmp3, tmp2, tid);
      if(tid % 32 >= off) {
        tmp2 += tmp3;
      }
    }
    if(tid < blockDim.x /32) {
      tmp[tid] = tmp2;
    }
  }
  __syncthreads();
  if(tid >= 32) {
    tmp1 += tmp[tid/32-1];
  }  
  d_in[threadIdx.x] = tmp1;

}

int main() {

 const int N = 1030;
 std::vector<int> h_in(N,0);
 std::vector<int> h_cpu_out(N,0);

 for(int i = 0; i < N; i++) {
   h_in[i] = 1;
 }
 int* d_in;
 hipMalloc((void**)&d_in, sizeof(int) * N);
 hipMemcpy(d_in, h_in.data(), sizeof(int)*N, hipMemcpyHostToDevice);
 
 scan <<< 2, 1024>>> (d_in, N);
 hipDeviceSynchronize();

 hipMemcpy(h_in.data(), d_in, sizeof(int)*N, hipMemcpyDeviceToHost);
 //std::exclusive_scan(h_in.begin(), h_in.end(), h_cpu_out.begin(), 0);
 for(int i = 0; i < N; i++) {
   std::cout << "i:" << i << ", val: " << h_in[i] << '\n';
 }
 hipFree(d_in);
  return 0;
}
